#include "hip/hip_runtime.h"
#include "HyperelasticityModule_NewtonMethod.h"
#include "Core/Utility.h"
#include "Framework/Framework/Node.h"
#include "Core/Algorithm/MatrixFunc.h"
#include "Kernel.h"

#include "Framework/Framework/Log.h"
#include "Core/Utility/Function1Pt.h"
#include "Core/Utility/math_utilities.h"

namespace PhysIKA
{
	template <typename Real, typename Coord>
	__global__ void computeDelta_vec(
		DeviceArray<Coord> vec1,
		DeviceArray<Coord> vec2,
		DeviceArray<Real> delta_norm)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= vec1.size()) return;

		delta_norm[pId] = (vec1[pId] - vec2[pId]).norm();
	}

	template <typename Real, typename Coord>
	__global__ void computeNorm_vec(
		DeviceArray<Coord> vec,
		DeviceArray<Real> norm)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= vec.size()) return;

		norm[pId] = vec[pId].norm();
	}

	template <typename Real, typename Coord>
	__global__ void computeRelativeError_vec(
		DeviceArray<Coord> vec1,
		DeviceArray<Coord> vec2,
		DeviceArray<Real> relative_error)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= vec1.size()) return;

		
	}

	template<typename TDataType>
	HyperelasticityModule_NewtonMethod<TDataType>::HyperelasticityModule_NewtonMethod()
		: ElasticityModule<TDataType>()
		, m_energyType(Linear)
	{
	}

	template <typename Coord, typename Matrix>
	COMM_FUNC Coord vec3_dot_mat3(
		Coord vec,
		Matrix mat) 
	{
		Coord result;
		result[0] = vec[0] * mat(0, 0) + vec[1] * mat(1, 0) + vec[2] * mat(2, 0);
		result[1] = vec[0] * mat(0, 1) + vec[1] * mat(1, 1) + vec[2] * mat(2, 1);
		result[2] = vec[0] * mat(0, 2) + vec[1] * mat(1, 2) + vec[2] * mat(2, 2);
		return result;
	}
	template <typename Coord, typename Matrix>
	COMM_FUNC Matrix vec3_outer_product_vec3(
		Coord vec1,
		Coord vec2,
		Matrix mat)
	{
		Matrix result;
		result(0, 0) += vec1[0] * vec2[0] ; result(0, 1) += vec1[0] * vec2[1] ; result(0, 2) += vec1[0] * vec2[2] ;
		result(1, 0) += vec1[1] * vec2[0] ; result(1, 1) += vec1[1] * vec2[1] ; result(1, 2) += vec1[1] * vec2[2] ;
		result(2, 0) += vec1[2] * vec2[0] ; result(2, 1) += vec1[2] * vec2[1] ; result(2, 2) += vec1[2] * vec2[2] ;
		return result;
	}
	template <typename Real, typename Matrix>
	COMM_FUNC Real mat3_double_product_mat3(
		Matrix mat1,
		Matrix mat2,
		Real type_arg)
	{
		return mat1(0, 0)*mat2(0, 0) + mat1(0, 1)*mat2(0, 1) + mat1(0, 2)*mat2(0, 2)
			+ mat1(1, 0)*mat2(1, 0) + mat1(1, 1)*mat2(1, 1) + mat1(1, 2)*mat2(1, 2)
			+ mat1(2, 0)*mat2(2, 0) + mat1(2, 1)*mat2(2, 1) + mat1(2, 2)*mat2(2, 2);
	}


	//**********compute total weight of each particle************************
	template <typename Real, typename Coord, typename NPair>
	__global__ void HM_ComputeTotalWeight_newton(
		DeviceArray<Coord> position,
		NeighborList<NPair> restShapes,
		DeviceArray<Real> totalWeight,
		Real horizon)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= position.size()) return;

		SmoothKernel<Real> kernSmooth;

		Coord rest_pos_i = restShapes.getElement(pId, 0).pos;
		int size_i = restShapes.getNeighborSize(pId);

		Real total_weight = Real(0);
		for (int ne = 1; ne < size_i; ne++)
		{
			NPair np_j = restShapes.getElement(pId, ne);
			int j = np_j.index;
			Coord rest_pos_j = np_j.pos;
			Real r = (rest_pos_i - rest_pos_j).norm();

			if (r > EPSILON)
			{
				Real weight = kernSmooth.Weight(r, horizon);
				total_weight += weight;
			}
		}

		totalWeight[pId] = total_weight;

	}

	// *************************  only update position **************************
	template <typename Coord>
	__global__ void HM_UpdatePosition_only(
		DeviceArray<Coord> position,
		DeviceArray<Coord> y_next)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= position.size()) return;

		position[pId] = y_next[pId];
	}

	template <typename Coord>
	__global__ void HM_UpdatePosition_delta_only(
		DeviceArray<Coord> position,
		DeviceArray<Coord> delta_y)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= position.size()) return;

		position[pId] = position[pId] + delta_y[pId];
	}

	template <typename Coord>
	__global__ void HM_UpdatePosition_Velocity(
		DeviceArray<Coord> position,
		DeviceArray<Coord> velocity,
		DeviceArray<Coord> y_next,
		DeviceArray<Coord> position_old,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= position.size()) return;

		position[pId] = y_next[pId];
		velocity[pId] += (position[pId] - position_old[pId]) / dt;
	}

	template <typename Coord>
	__global__ void HM_UpdateVelocity_only(
		DeviceArray<Coord> position,
		DeviceArray<Coord> velocity,
		DeviceArray<Coord> position_old,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= position.size()) return;

		velocity[pId] += (position[pId] - position_old[pId]) / dt;
	}


	template <typename Real, typename Coord, typename Matrix>
	__global__ void HM_ComputeTotalEnergy_Linear(
		DeviceArray<Real> energy_i,
		DeviceArray<Coord> position,
		DeviceArray<Coord> position_old,
		DeviceArray<Matrix> F,
		Real mu,
		Real lambda,
		Real mass,
		Real volume,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= position.size()) return;

		Matrix F_i = F[pId];
		Matrix epsilon = 0.5*(F_i + F_i.transpose()) - Matrix::identityMatrix();
		Real elasticity_energy_density_i = mu * mat3_double_product_mat3(epsilon, epsilon, mass) + 0.5*lambda*epsilon.trace()*epsilon.trace();

		energy_i[pId] = 0.5*mass * (position[pId]-position_old[pId]).normSquared()/(dt*dt) + volume* elasticity_energy_density_i;
	}
	template <typename Real, typename Coord, typename Matrix>
	__global__ void HM_ComputeTotalEnergy_StVK(
		DeviceArray<Real> energy_i,
		DeviceArray<Coord> position,
		DeviceArray<Coord> position_old,
		DeviceArray<Matrix> F,
		Real mass,
		Real volume,
		Real mu,
		Real lambda,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= position.size()) return;

		Matrix F_i = F[pId];
		Matrix E_i = 0.5*(F_i.transpose()*F_i - Matrix::identityMatrix());
		Real elasticity_energy_density_i = mu * mat3_double_product_mat3(E_i, E_i, mass) + 0.5*lambda*E_i.trace()*E_i.trace();

		energy_i[pId] = 0.5*mass * (position[pId] - position_old[pId]).normSquared() / (dt*dt) + volume * elasticity_energy_density_i;
	}


	template <typename Real, typename Coord, typename Matrix>
	COMM_FUNC Matrix HM_ComputeHessianMatrix_LinearEnergy(
		int index_energy,
		int index_i,
		int index_j,
		Coord dx_ij,
		Coord dx_ji,
		Coord delta_x_i,
		Coord delta_x_j,
		Real horizon,
		Real mu, Real lambda,
		Real mass, Real volume,
		Real weight_ij,
		Real weight_ji,
		Matrix identityMat)
	{
		if (index_energy == index_i) {
			if (index_i == index_j) {
				Matrix result(0.0);
				result = volume * volume * (
					mu*(delta_x_i.dot(delta_x_i))*Matrix::identityMatrix() 
					+(mu + lambda)*vec3_outer_product_vec3(delta_x_i, delta_x_i, Matrix::identityMatrix() ));
				return result;
			}
			else {
				Matrix result(0.0);
				
				result = weight_ij * volume*volume*( 
					mu*dx_ji.dot(delta_x_i)*Matrix::identityMatrix() 
					+ mu*vec3_outer_product_vec3(dx_ji, delta_x_i, Matrix::identityMatrix())
					+ lambda*vec3_outer_product_vec3(delta_x_i, dx_ji, Matrix::identityMatrix()) );
		
				return result;
			}
		}
		else if(index_energy == index_j){
			Matrix result(0.0);

			if (index_i == index_j) {
				result = weight_ji * weight_ji * volume*volume*(
					mu*dx_ij.dot(dx_ij)*Matrix::identityMatrix()
					+ (mu + lambda) * vec3_outer_product_vec3(dx_ij, dx_ij, Matrix::identityMatrix()) );
				
				return result;
			}
			else {
				result = weight_ji * volume*volume*(
					mu*delta_x_j.dot(dx_ij)*Matrix::identityMatrix()
					+ mu * vec3_outer_product_vec3(delta_x_j, dx_ij, Matrix::identityMatrix())
					+ lambda * vec3_outer_product_vec3(dx_ij, delta_x_j, Matrix::identityMatrix()));
			
				return result;
			}
		}
		else {
			return Matrix(0.0);
		}
		
	}


	template <typename Real, typename Coord, typename Matrix, typename NPair>
	__global__ void HM_ComputeSourceTerm_Linear(
		DeviceArray<Coord> sourceItems,
		DeviceArray<Matrix> inverseK,
		DeviceArray<Matrix> stressTensors,
		DeviceArray<Coord> position_old,
		DeviceArray<Coord> y_current,
		DeviceArray<Coord> Sum_delta_x,
		NeighborList<NPair> restShapes,
		Real horizon,
		Real mu, Real lambda,
		Real mass, Real volume, Real dt,
		Real weightScale) 
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= stressTensors.size()) return;

		Coord delta_x_i = Sum_delta_x[pId];
		Coord y_i = y_current[pId];
		Matrix invK_i = inverseK[pId];
		int index_i = pId;
		Coord rest_pos_i = restShapes.getElement(pId, 0).pos;
		int size_i = restShapes.getNeighborSize(pId);

		Coord energy_gradient_i = Coord(0.0);
		energy_gradient_i += mass * (y_current[pId] - position_old[pId]) / (dt*dt);
		Coord linear_gradient_Wi_i = volume * (stressTensors[index_i]*delta_x_i) ;
		energy_gradient_i += volume * linear_gradient_Wi_i; // not finished

		Coord b_i = Coord(0.0);

		SmoothKernel<Real> kernSmooth;

		for (int ne = 1; ne < size_i; ne++)
		{
			NPair np_j = restShapes.getElement(pId, ne);
			int index_j = np_j.index;
			Coord rest_pos_j = np_j.pos;
			Real r = (rest_pos_i - rest_pos_j).norm();

			Coord y_j = y_current[index_j];

			if (r > EPSILON)
			{
				Real weight = kernSmooth.Weight(r, horizon);
				weight = weight / weightScale;

				Matrix invK_j = inverseK[index_j];
				Coord delta_x_j = Sum_delta_x[index_j];
				Coord dx_ji = vec3_dot_mat3((rest_pos_j - rest_pos_i) / (horizon*horizon), invK_i);
				Coord dx_ij = vec3_dot_mat3((rest_pos_i - rest_pos_j) / (horizon*horizon), invK_j);
				
				Coord linear_gradient_Wj_i = weight * volume *(stressTensors[index_j] * dx_ij);
				energy_gradient_i += volume * linear_gradient_Wj_i;
			}
		}
		b_i = -energy_gradient_i;

		sourceItems[pId] = b_i;
	}

	template <typename Real, typename Coord, typename Matrix>
	COMM_FUNC Matrix HM_ComputeHessianMatrix_StVKEnergy(
		int index_energy,
		int index_i,
		int index_j,
		Coord dx_ij,
		Coord dx_ji,
		Coord delta_x_i,
		Coord delta_x_j,
		Real horizon,
		Real mu, Real lambda,
		Real mass, Real volume,
		Real weight_ij,
		Real weight_ji,
		Matrix F,
		Matrix E)
	{
		if (index_energy == index_i) {
			if (index_i == index_j) {
				Matrix result(0.0);
				result = volume * volume * (
					2*mu*(delta_x_i.dot( E*delta_x_i ))*Matrix::identityMatrix()
					+ mu*vec3_outer_product_vec3(F * delta_x_i, F * delta_x_i, Matrix::identityMatrix()) 
					+ mu * delta_x_i.dot(delta_x_i)*( F * F.transpose() )
					+ lambda* vec3_outer_product_vec3(F * delta_x_i, F * delta_x_i, Matrix::identityMatrix())
					+ lambda * E.trace() * delta_x_i.dot(delta_x_i) * Matrix::identityMatrix()
					);
				return result;
			}
			else {
				Matrix result(0.0);

				result = weight_ij * volume * volume * (
					2 * mu*(dx_ji.dot(E*delta_x_i))*Matrix::identityMatrix()
					+ mu * vec3_outer_product_vec3(F * dx_ji, F * delta_x_i, Matrix::identityMatrix())
					+ mu * dx_ji.dot(delta_x_i)*(F * F.transpose())
					+ lambda * vec3_outer_product_vec3(F * delta_x_i, F * dx_ji, Matrix::identityMatrix())
					+ lambda * E.trace() * dx_ji.dot(delta_x_i) * Matrix::identityMatrix()
					);

				return result;
			}
		}
		else if (index_energy == index_j) {
			Matrix result(0.0);

			if (index_i == index_j) {
				result = weight_ji * weight_ji * volume * volume * (
					2 * mu*(dx_ij.dot(E*dx_ij))*Matrix::identityMatrix()
					+ mu * vec3_outer_product_vec3(F * dx_ij, F * dx_ij, Matrix::identityMatrix())
					+ mu * dx_ij.dot(dx_ij)*(F * F.transpose())
					+ lambda * vec3_outer_product_vec3(F * dx_ij, F * dx_ij, Matrix::identityMatrix())
					+ lambda * E.trace() * dx_ij.dot(dx_ij) * Matrix::identityMatrix()
					);

				return result;
			}
			else {
				result = weight_ji * volume * volume * (
					2 * mu*(delta_x_j.dot(E*dx_ij))*Matrix::identityMatrix()
					+ mu * vec3_outer_product_vec3(F * delta_x_j, F * dx_ij, Matrix::identityMatrix())
					+ mu * delta_x_j.dot(dx_ij)*(F * F.transpose())
					+ lambda * vec3_outer_product_vec3(F * dx_ij, F * delta_x_j, Matrix::identityMatrix())
					+ lambda * E.trace() * delta_x_j.dot(dx_ij) * Matrix::identityMatrix()
					);

				return result;
			}
		}
		else {
			return Matrix(0.0);
		}

	}

	template <typename Real, typename Coord, typename Matrix, typename NPair>
	__global__ void HM_ComputeSourceTerm_StVK(
		DeviceArray<Coord> sourceItems,
		DeviceArray<Matrix> F,
		DeviceArray<Matrix> inverseK,
		DeviceArray<Matrix> stressTensors,
		DeviceArray<Coord> position_old,
		DeviceArray<Coord> y_current,
		DeviceArray<Coord> Sum_delta_x,
		NeighborList<NPair> restShapes,
		Real horizon,
		Real mu, Real lambda,
		Real mass, Real volume, Real dt,
		Real weightScale)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= stressTensors.size()) return;

		Coord delta_x_i = Sum_delta_x[pId];
		Coord y_i = y_current[pId];
		Matrix invK_i = inverseK[pId];
		int index_i = pId;
		Coord rest_pos_i = restShapes.getElement(pId, 0).pos;
		int size_i = restShapes.getNeighborSize(pId);

		Coord energy_gradient_i = Coord(0.0);
		energy_gradient_i += mass * (y_current[pId] - position_old[pId]) / (dt*dt);
		Coord linear_gradient_Wi_i = volume * (stressTensors[index_i] * delta_x_i);
		energy_gradient_i += volume * linear_gradient_Wi_i; // not finished

		Coord b_i = Coord(0.0);

		SmoothKernel<Real> kernSmooth;

		for (int ne = 1; ne < size_i; ne++)
		{
			NPair np_j = restShapes.getElement(pId, ne);
			int index_j = np_j.index;
			Coord rest_pos_j = np_j.pos;
			Real r = (rest_pos_i - rest_pos_j).norm();

			Coord y_j = y_current[index_j];

			if (r > EPSILON)
			{
				Real weight = kernSmooth.Weight(r, horizon);
				weight = weight / weightScale;

				Matrix invK_j = inverseK[index_j];
				Coord delta_x_j = Sum_delta_x[index_j];
				Coord dx_ji = vec3_dot_mat3((rest_pos_j - rest_pos_i) / (horizon*horizon), invK_i);
				Coord dx_ij = vec3_dot_mat3((rest_pos_i - rest_pos_j) / (horizon*horizon), invK_j);

				Coord linear_gradient_Wj_i = weight *volume* (stressTensors[index_j] * dx_ij);
				energy_gradient_i += volume * linear_gradient_Wj_i;
			}
		}

		b_i = -energy_gradient_i;

		sourceItems[pId] = b_i;
	}

	// these deformation gradients are mat3x3, may be singular
	template <typename Real, typename Coord, typename Matrix, typename NPair>
	__global__ void HM_ComputeFandSdx(
		DeviceArray<Matrix> inverseK,
		DeviceArray<Matrix> F,
		DeviceArray<Coord> Sum_delta_x,
		DeviceArray<Coord> position,
		NeighborList<NPair> restShapes,
		Real horizon,
		Real weightScale)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= position.size()) return;

		SmoothKernel<Real> kernSmooth;

		Coord rest_pos_i = restShapes.getElement(pId, 0).pos;
		int size_i = restShapes.getNeighborSize(pId);

		Matrix matL_i(0);
		Matrix matK_i(0);
		Coord Delta_x = Coord(0.0);

		for (int ne = 1; ne < size_i; ne++)
		{
			NPair np_j = restShapes.getElement(pId, ne);
			int j = np_j.index;
			Coord rest_pos_j = np_j.pos;
			Real r = (rest_pos_i - rest_pos_j).norm();

			if (r > EPSILON)
			{
				Real weight = kernSmooth.Weight(r, horizon);
				weight = weight / weightScale;

				Coord p = (position[j] - position[pId]) / horizon;
				Coord q = (rest_pos_j - rest_pos_i) / horizon;

				Delta_x += weight * (rest_pos_i - rest_pos_j)/(horizon*horizon);

				matL_i(0, 0) += p[0] * q[0] * weight; matL_i(0, 1) += p[0] * q[1] * weight; matL_i(0, 2) += p[0] * q[2] * weight;
				matL_i(1, 0) += p[1] * q[0] * weight; matL_i(1, 1) += p[1] * q[1] * weight; matL_i(1, 2) += p[1] * q[2] * weight;
				matL_i(2, 0) += p[2] * q[0] * weight; matL_i(2, 1) += p[2] * q[1] * weight; matL_i(2, 2) += p[2] * q[2] * weight;

				matK_i(0, 0) += q[0] * q[0] * weight; matK_i(0, 1) += q[0] * q[1] * weight; matK_i(0, 2) += q[0] * q[2] * weight;
				matK_i(1, 0) += q[1] * q[0] * weight; matK_i(1, 1) += q[1] * q[1] * weight; matK_i(1, 2) += q[1] * q[2] * weight;
				matK_i(2, 0) += q[2] * q[0] * weight; matK_i(2, 1) += q[2] * q[1] * weight; matK_i(2, 2) += q[2] * q[2] * weight;

			}
		}

		Matrix R, U, D, V;
		polarDecomposition(matK_i, R, U, D, V);
		//	getSVDmatrix(matK_i, &U, &D, &V);

		Real threshold = 0.0001f*horizon;
		D(0, 0) = D(0, 0) > threshold ? 1.0 / D(0, 0) : 1.0;
		D(1, 1) = D(1, 1) > threshold ? 1.0 / D(1, 1) : 1.0;
		D(2, 2) = D(2, 2) > threshold ? 1.0 / D(2, 2) : 1.0;

		Matrix inv_mat_K = V * D*U.transpose();

		inverseK[pId] = inv_mat_K;

		Delta_x = vec3_dot_mat3(Delta_x, inv_mat_K);
		Sum_delta_x[pId] = Delta_x;
		F[pId] = matL_i * inv_mat_K;
	}

	template <typename Real, typename Matrix>
	__global__ void HM_ComputeFirstPiolaKirchhoff_Linear(
		DeviceArray<Matrix> stressTensor,
		DeviceArray<Matrix> F,
		Real mu,
		Real lambda)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= F.size()) return;

		Matrix F_i = F[pId];

		// find infinitesimal strain tensor epsilon = 1/2(F + F^T) - I
		Matrix epsilon = 0.5*(F_i.transpose() + F_i) - Matrix::identityMatrix();
		// find first Piola-Kirchhoff matix; Linear material
		stressTensor[pId] = 2 * mu * epsilon + lambda * epsilon.trace() * Matrix::identityMatrix();

	}

	template <typename Real, typename Matrix>
	__global__ void HM_ComputeFirstPiolaKirchhoff_StVK(
		DeviceArray<Matrix> stressTensor,
		DeviceArray<Matrix> F,
		Real mu,
		Real lambda)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= F.size()) return;

		Matrix F_i = F[pId];

		// find strain tensor E = 1/2(F^T * F - I)
		Matrix E = 0.5*(F_i.transpose() * F_i - Matrix::identityMatrix());
		// find first Piola-Kirchhoff matix; StVK material
		stressTensor[pId] = F_i * (2 * mu * E + lambda * E.trace() * Matrix::identityMatrix());
	}


	template <typename Real, typename Coord, typename Matrix, typename NPair>
	__global__ void HM_JacobiStep_Linear(
		DeviceArray<Coord> delta_y_new,
		DeviceArray<Coord> delta_y_old,
		DeviceArray<Coord> sourceItems,
		DeviceArray<Matrix> inverseK,
		DeviceArray<Coord> Sum_delta_x,
		NeighborList<NPair> restShapes,
		Real horizon,
		Real mu, Real lambda,
		Real mass,
		Real volume,
		Real dt,
		Real weightScale)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= delta_y_old.size()) return;

		Coord totalSource_i = sourceItems[pId];
		// not finished

		Coord delta_x_i = Sum_delta_x[pId];

		Matrix invK_i = inverseK[pId];
		int index_i = pId;
		int size_i = restShapes.getNeighborSize(pId);
		Coord rest_pos_i = restShapes.getElement(pId, 0).pos;

		Matrix partial_Wi_i_i = HM_ComputeHessianMatrix_LinearEnergy(
			index_i, index_i, index_i,
			Coord(0.0), Coord(0.0),
			delta_x_i, delta_x_i,
			horizon,
			mu, lambda,
			mass, volume,
			Real(0.0), Real(0.0),
			Matrix::identityMatrix());
		Matrix hessian_i_i = (mass / (dt*dt)) * Matrix::identityMatrix() + volume * partial_Wi_i_i;
		// hessian_i_i not finished

		SmoothKernel<Real> kernSmooth;
		for (int ne = 1; ne < size_i; ne++)
		{
			NPair np_j = restShapes.getElement(pId, ne);
			int index_j = np_j.index;
			Coord rest_pos_j = np_j.pos;
			Coord delta_y_j = delta_y_old[index_j];
			Real r = (rest_pos_j - rest_pos_i).norm();

			if (r > EPSILON)
			{
				Real weight = kernSmooth.Weight(r, horizon);
				weight = weight / weightScale;

				Matrix invK_j = inverseK[index_j];
				Coord delta_x_j = Sum_delta_x[index_j];
				Coord dx_ji = vec3_dot_mat3((rest_pos_j - rest_pos_i) / (horizon*horizon), invK_i);
				Coord dx_ij = vec3_dot_mat3((rest_pos_i - rest_pos_j) / (horizon*horizon), invK_j);

				Matrix hessian_Wj_i_i = HM_ComputeHessianMatrix_LinearEnergy(
					index_j, index_i, index_i,
					Coord(0.0), Coord(0.0),
					delta_x_i, delta_x_i,
					horizon,
					mu, lambda,
					mass, volume,
					weight, weight,
					Matrix::identityMatrix());

				hessian_i_i += volume * hessian_Wj_i_i;

				Matrix partial_Wi_i_j = HM_ComputeHessianMatrix_LinearEnergy(
					index_i, index_i, index_j,
					dx_ij, dx_ji,
					delta_x_i, delta_x_j,
					horizon,
					mu, lambda,
					mass, volume,
					weight, weight,
					Matrix::identityMatrix());
				Matrix partial_Wj_i_j = HM_ComputeHessianMatrix_LinearEnergy(
					index_j, index_i, index_j,
					dx_ij, dx_ji,
					delta_x_i, delta_x_j,
					horizon,
					mu, lambda,
					mass, volume,
					weight, weight,
					Matrix::identityMatrix());

				totalSource_i -= (volume*partial_Wi_i_j + volume * partial_Wj_i_j)*delta_y_j;
			}
		}

		delta_y_new[pId] = hessian_i_i.inverse()*totalSource_i;
	}

	template <typename Real, typename Coord, typename Matrix, typename NPair>
	__global__ void HM_JacobiStep_StVK(
		DeviceArray<Coord> delta_y_new,
		DeviceArray<Coord> delta_y_old,
		DeviceArray<Coord> sourceItems,
		DeviceArray<Matrix> F,
		DeviceArray<Matrix> inverseK,
		DeviceArray<Coord> Sum_delta_x,
		NeighborList<NPair> restShapes,
		Real horizon,
		Real mu, Real lambda,
		Real mass,
		Real volume,
		Real dt,
		Real weightScale)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= delta_y_old.size()) return;

		Coord totalSource_i = sourceItems[pId];
		// not finished

		Coord delta_x_i = Sum_delta_x[pId];

		Matrix invK_i = inverseK[pId];
		int index_i = pId;
		int size_i = restShapes.getNeighborSize(pId);
		Coord rest_pos_i = restShapes.getElement(pId, 0).pos;

		Matrix F_i = F[pId];
		Matrix E_i = 0.5*(F_i.transpose() * F_i - Matrix::identityMatrix());
		Matrix partial_Wi_i_i = HM_ComputeHessianMatrix_StVKEnergy(
			index_i, index_i, index_i,
			Coord(0.0), Coord(0.0),
			delta_x_i, delta_x_i,
			horizon,
			mu, lambda,
			mass, volume,
			Real(0.0), Real(0.0),
			F_i, E_i);
		Matrix hessian_i_i = (mass / (dt*dt)) * Matrix::identityMatrix() + volume * partial_Wi_i_i;
		// hessian_i_i not finished

		SmoothKernel<Real> kernSmooth;
		for (int ne = 1; ne < size_i; ne++)
		{
			NPair np_j = restShapes.getElement(pId, ne);
			int index_j = np_j.index;
			Coord rest_pos_j = np_j.pos;
			Coord delta_y_j = delta_y_old[index_j];
			Real r = (rest_pos_j - rest_pos_i).norm();

			if (r > EPSILON)
			{
				Real weight = kernSmooth.Weight(r, horizon);
				weight = weight / weightScale;

				Matrix invK_j = inverseK[index_j];
				Coord delta_x_j = Sum_delta_x[index_j];
				Coord dx_ji = vec3_dot_mat3((rest_pos_j - rest_pos_i) / (horizon*horizon), invK_i);
				Coord dx_ij = vec3_dot_mat3((rest_pos_i - rest_pos_j) / (horizon*horizon), invK_j);

				Matrix F_j = F[index_j];
				Matrix E_j = 0.5*(F_j.transpose() * F_j - Matrix::identityMatrix());
				Matrix hessian_Wj_i_i = HM_ComputeHessianMatrix_StVKEnergy(
					index_j, index_i, index_i,
					Coord(0.0), Coord(0.0),
					delta_x_i, delta_x_i,
					horizon,
					mu, lambda,
					mass, volume,
					weight, weight,
					F_j, E_j);

				hessian_i_i += volume * hessian_Wj_i_i;

				Matrix partial_Wi_i_j = HM_ComputeHessianMatrix_StVKEnergy(
					index_i, index_i, index_j,
					dx_ij, dx_ji,
					delta_x_i, delta_x_j,
					horizon,
					mu, lambda,
					mass, volume,
					weight, weight,
					F_i, E_i);
				Matrix partial_Wj_i_j = HM_ComputeHessianMatrix_StVKEnergy(
					index_j, index_i, index_j,
					dx_ij, dx_ji,
					delta_x_i, delta_x_j,
					horizon,
					mu, lambda,
					mass, volume,
					weight, weight,
					F_j, E_j);

				totalSource_i -= (volume*partial_Wi_i_j + volume * partial_Wj_i_j)*delta_y_j;
			}
		}

		delta_y_new[pId] = hessian_i_i.inverse()*totalSource_i;
	}

	template<typename TDataType>
	bool HyperelasticityModule_NewtonMethod<TDataType>::initializeImpl()
	{
		m_position_old.resize(this->inPosition()->getElementCount());
		m_F.resize(this->inPosition()->getElementCount());
		m_invK.resize(this->inPosition()->getElementCount());
		m_firstPiolaKirchhoffStress.resize(this->inPosition()->getElementCount());

		m_totalWeight.resize(this->inPosition()->getElementCount());
		m_Sum_delta_x.resize(this->inPosition()->getElementCount());
		m_source_items.resize(this->inPosition()->getElementCount() );

		debug_pos_isNaN = false;
		debug_v_isNaN = false;
		debug_invL_isNaN = false;
		debug_F_isNaN = false;
		debug_invF_isNaN = false;
		debug_Piola_isNaN = true;

		return ElasticityModule::initializeImpl();
	}


	template<typename TDataType>
	void HyperelasticityModule_NewtonMethod<TDataType>::solveElasticity()
	{
		typedef typename TDataType::Real Real;
		typedef typename TDataType::Coord Coord;
		typedef typename TDataType::Matrix Matrix;


		int numOfParticles = this->inPosition()->getElementCount();
		uint pDims = cudaGridSize(numOfParticles, BLOCK_SIZE);
		HM_ComputeTotalWeight_newton << <pDims, BLOCK_SIZE >> > (
			this->inPosition()->getValue(),
			this->m_restShape.getValue(),
			this->m_totalWeight,
			this->inHorizon()->getValue());
		cuSynchronize();

		{
			Reduction<Real>* pReduction = Reduction<Real>::Create(numOfParticles);
			Real max_totalWeight = pReduction->maximum(this->m_totalWeight.getDataPtr(), numOfParticles);
			printf("Max total weight: %f \n", max_totalWeight);
		}

		solveElasticity_NewtonMethod();

	}

	template<typename TDataType>
	void HyperelasticityModule_NewtonMethod<TDataType>::solveElasticity_NewtonMethod()
	{
		typedef typename TDataType::Real Real;
		typedef typename TDataType::Coord Coord;
		typedef typename TDataType::Matrix Matrix;

		int numOfParticles = this->inPosition()->getElementCount();
		uint pDims = cudaGridSize(numOfParticles, BLOCK_SIZE);

		this->m_displacement.reset();
		this->m_weights.reset();

		Log::sendMessage(Log::User, "solver start!!!");

		// mass and volume are set 1.0, (need modified) 
		Real mass = 1.0;
		Real volume = 1.0;

		// initialize y_now, y_next_iter
		DeviceArray<Coord> delta_y_pre(numOfParticles);
		DeviceArray<Coord> delta_y_next(numOfParticles);

		delta_y_pre.reset();
		delta_y_next.reset();
		Function1Pt::copy(m_position_old, this->inPosition()->getValue());

		// do Jacobi method Loop
		bool newton_convergeFlag = false; // outer loop(newton method) converge or not
		bool jacobi_convergeFlag = false; // inner loop(jacobi method) converge or not
		int newton_iteNum = 0;
		int jacobi_iteNum = 0;
		int jacobi_total_iteNum = 0;
		int	newton_maxIterations = 50;
		int jacobi_maxIterations = 200;
		double converge_threshold = 0.001f*this->inHorizon()->getValue();
		double relative_error_threshold = 0.001;

		double newton_first_delta = 0.0;
		double jacobi_first_delta = 0.0;

		double last_state_energy = DBL_MAX;

		int energy_rise_times = 0;

		for (newton_iteNum = 0; newton_iteNum < newton_maxIterations; ++newton_iteNum) { // newton method loop: H*y_{k+1} = H*y_{k} + gradient of f 

			delta_y_pre.reset();
			delta_y_next.reset();

			HM_ComputeFandSdx << <pDims, BLOCK_SIZE >> > (
				m_invK,
				m_F,
				m_Sum_delta_x,
				this->inPosition()->getValue(),
				this->m_restShape.getValue(),
				this->inHorizon()->getValue(),
				this->weightScale);
			cuSynchronize();

			{
				DeviceArray<Real> energy_particles(numOfParticles);
				HM_ComputeTotalEnergy_Linear << <pDims, BLOCK_SIZE >> > (
					energy_particles,
					this->inPosition()->getValue(),
					m_position_old,
					m_F,
					this->m_mu.getValue(),
					this->m_lambda.getValue(),
					mass, volume,
					this->getParent()->getDt() );
				cuSynchronize();

				Reduction<Real>* pReduction = Reduction<Real>::Create(numOfParticles);
				Real current_energy = pReduction->accumulate(energy_particles.getDataPtr(), numOfParticles);
				energy_particles.release();

				if (current_energy >= last_state_energy) {
					energy_rise_times++;
				}
				last_state_energy = current_energy;
			}

			HM_ComputeFirstPiolaKirchhoff_Linear << <pDims, BLOCK_SIZE >> > (
				m_firstPiolaKirchhoffStress,
				m_F,
				this->m_mu.getValue(),
				this->m_lambda.getValue());
			cuSynchronize();

			HM_ComputeSourceTerm_Linear << <pDims, BLOCK_SIZE >> > (
				m_source_items,
				m_invK,
				m_firstPiolaKirchhoffStress,
				m_position_old,
				this->inPosition()->getValue(),
				m_Sum_delta_x,
				this->m_restShape.getValue(),
				this->inHorizon()->getValue(),
				this->m_mu.getValue(),
				this->m_lambda.getValue(),
				mass, volume, this->getParent()->getDt(),
				this->weightScale);
			cuSynchronize();

			jacobi_convergeFlag = false;
			for (jacobi_iteNum = 0; jacobi_iteNum < jacobi_maxIterations; ++jacobi_iteNum) { // jacobi method loop

				HM_JacobiStep_Linear << <pDims, BLOCK_SIZE >> > (
					delta_y_next,
					delta_y_pre,
					m_source_items,
					m_invK,
					m_Sum_delta_x,
					this->m_restShape.getValue(),
					this->inHorizon()->getValue(),
					this->m_mu.getValue(),
					this->m_lambda.getValue(),
					mass, volume,
					this->getParent()->getDt(),
					this->weightScale);
				cuSynchronize();

				{
					Reduction<Real>* pReduction = Reduction<Real>::Create(numOfParticles);
					DeviceArray<Real> Delta_y_norm(numOfParticles);
					computeNorm_vec << <pDims, BLOCK_SIZE >> >(delta_y_next, Delta_y_norm);
					cuSynchronize();

					Real max_delta = pReduction->maximum(Delta_y_norm.getDataPtr(), numOfParticles);
					Delta_y_norm.release();

					if (jacobi_iteNum == 0) {
						jacobi_first_delta = max_delta;
						if (jacobi_first_delta == 0.0) { jacobi_convergeFlag = true; }
					}
					else {
						if ( (max_delta/jacobi_first_delta) < relative_error_threshold) { jacobi_convergeFlag = true; }
					}
				}

				Function1Pt::copy(delta_y_pre, delta_y_next);
				if (jacobi_convergeFlag) { break; }
			}

			if (jacobi_iteNum < jacobi_maxIterations) { jacobi_iteNum++; }
			jacobi_total_iteNum += jacobi_iteNum;

			{
				Reduction<Real>* pReduction = Reduction<Real>::Create(numOfParticles);
				DeviceArray<Real> Delta_y_norm(numOfParticles);

				computeNorm_vec << <pDims, BLOCK_SIZE >> >(delta_y_next, Delta_y_norm);
				cuSynchronize();

				Real max_delta = pReduction->maximum(Delta_y_norm.getDataPtr(), numOfParticles);
				Delta_y_norm.release();

				if (newton_iteNum == 0) {
					newton_first_delta = max_delta;
					if (newton_first_delta == 0.0) { newton_convergeFlag = true; }
				}
				else {
					if ( (max_delta/newton_first_delta) < relative_error_threshold) { newton_convergeFlag = true; }
				}
			}

			HM_UpdatePosition_delta_only << <pDims, BLOCK_SIZE >> > (
				this->inPosition()->getValue(),
				delta_y_next);
			cuSynchronize();

			if (newton_convergeFlag) { break; }
		}

		HM_UpdateVelocity_only << <pDims, BLOCK_SIZE >> > (
			this->inPosition()->getValue(),
			this->inVelocity()->getValue(),
			m_position_old,
			this->getParent()->getDt());
		cuSynchronize();

		delta_y_pre.release();
		delta_y_next.release();

		if (newton_iteNum < newton_maxIterations) { newton_iteNum++; }
		printf("newton ite num: %d \n jacobi ave_ite num: %f \n", newton_iteNum, double(jacobi_total_iteNum) / double(newton_iteNum));
		printf("energy rise times: %d\n", energy_rise_times);
		if (jacobi_convergeFlag) { printf("jacobi converge!"); }
		if (newton_convergeFlag) { printf("newton converge!"); }
	}

	template<typename TDataType>
	void HyperelasticityModule_NewtonMethod<TDataType>::solveElasticity_NewtonMethod_StVK()
	{
		typedef typename TDataType::Real Real;
		typedef typename TDataType::Coord Coord;
		typedef typename TDataType::Matrix Matrix;

		int numOfParticles = this->inPosition()->getElementCount();
		uint pDims = cudaGridSize(numOfParticles, BLOCK_SIZE);

		this->m_displacement.reset();
		this->m_weights.reset();

		Log::sendMessage(Log::User, "solver start!!!");

		// mass and volume are set 1.0, (need modified) 
		Real mass = 1.0;
		Real volume = 1.0;

		// initialize y_now, y_next_iter
		DeviceArray<Coord> delta_y_pre(numOfParticles);
		DeviceArray<Coord> delta_y_next(numOfParticles);

		delta_y_pre.reset();
		delta_y_next.reset();
		Function1Pt::copy(m_position_old, this->inPosition()->getValue());

		// do Jacobi method Loop
		bool newton_convergeFlag = false; // outer loop(newton method) converge or not
		bool jacobi_convergeFlag = false; // inner loop(jacobi method) converge or not
		int newton_iteNum = 0;
		int jacobi_iteNum = 0;
		int jacobi_total_iteNum = 0;
		int	newton_maxIterations = 50;
		int jacobi_maxIterations = 200;
		double converge_threshold = 0.001f*this->inHorizon()->getValue();
		double relative_error_threshold = 0.001;

		double newton_first_delta = 0.0;
		double jacobi_first_delta = 0.0;

		for (newton_iteNum = 0; newton_iteNum < newton_maxIterations; ++newton_iteNum) { // newton method loop: H*y_{k+1} = H*y_{k} + gradient of f 

			delta_y_pre.reset();
			delta_y_next.reset();

			HM_ComputeFandSdx << <pDims, BLOCK_SIZE >> > (
				m_invK,
				m_F,
				m_Sum_delta_x,
				this->inPosition()->getValue(),
				this->m_restShape.getValue(),
				this->inHorizon()->getValue(),
				this->weightScale);
			cuSynchronize();

			HM_ComputeFirstPiolaKirchhoff_StVK << <pDims, BLOCK_SIZE >> > (
				m_firstPiolaKirchhoffStress,
				m_F,
				this->m_mu.getValue(),
				this->m_lambda.getValue());
			cuSynchronize();

			HM_ComputeSourceTerm_StVK << <pDims, BLOCK_SIZE >> > (
				m_source_items,
				m_F,
				m_invK,
				m_firstPiolaKirchhoffStress,
				m_position_old,
				this->inPosition()->getValue(),
				m_Sum_delta_x,
				this->m_restShape.getValue(),
				this->inHorizon()->getValue(),
				this->m_mu.getValue(),
				this->m_lambda.getValue(),
				mass, volume, this->getParent()->getDt(),
				this->weightScale);
			cuSynchronize();

			jacobi_convergeFlag = false;
			for (jacobi_iteNum = 0; jacobi_iteNum < jacobi_maxIterations; ++jacobi_iteNum) { // jacobi method loop

				HM_JacobiStep_StVK << <pDims, BLOCK_SIZE >> > (
					delta_y_next,
					delta_y_pre,
					m_source_items,
					m_F,
					m_invK,
					m_Sum_delta_x,
					this->m_restShape.getValue(),
					this->inHorizon()->getValue(),
					this->m_mu.getValue(),
					this->m_lambda.getValue(),
					mass, volume,
					this->getParent()->getDt(),
					this->weightScale);
				cuSynchronize();

				{
					Reduction<Real>* pReduction = Reduction<Real>::Create(numOfParticles);
					DeviceArray<Real> Delta_y_norm(numOfParticles);
					computeNorm_vec << <pDims, BLOCK_SIZE >> >(delta_y_next, Delta_y_norm);
					cuSynchronize();

					Real max_delta = pReduction->maximum(Delta_y_norm.getDataPtr(), numOfParticles);
					Delta_y_norm.release();

					if (jacobi_iteNum == 0) {
						jacobi_first_delta = max_delta;
						if (jacobi_first_delta == 0.0) { jacobi_convergeFlag = true; }
					}
					else {
						if (max_delta / jacobi_first_delta < relative_error_threshold) { jacobi_convergeFlag = true; }
					}
				}

				Function1Pt::copy(delta_y_pre, delta_y_next);
				if (jacobi_convergeFlag) { break; }
			}

			if (jacobi_iteNum < jacobi_maxIterations) { jacobi_iteNum++; }
			jacobi_total_iteNum += jacobi_iteNum;

			{
				Reduction<Real>* pReduction = Reduction<Real>::Create(numOfParticles);
				DeviceArray<Real> Delta_y_norm(numOfParticles);

				computeNorm_vec << <pDims, BLOCK_SIZE >> >(delta_y_next, Delta_y_norm);
				cuSynchronize();

				Real max_delta = pReduction->maximum(Delta_y_norm.getDataPtr(), numOfParticles);
				Delta_y_norm.release();

				if (newton_iteNum == 0) {
					newton_first_delta = max_delta;
					if (newton_first_delta == 0.0) { newton_convergeFlag = true; }
				}
				else {
					if (max_delta / newton_first_delta < relative_error_threshold) { newton_convergeFlag = true; }
				}
			}

			HM_UpdatePosition_delta_only << <pDims, BLOCK_SIZE >> > (
				this->inPosition()->getValue(),
				delta_y_next);
			cuSynchronize();

			if (newton_convergeFlag) { break; }
		}

		HM_UpdateVelocity_only << <pDims, BLOCK_SIZE >> > (
			this->inPosition()->getValue(),
			this->inPosition()->getValue(),
			m_position_old,
			this->getParent()->getDt());
		cuSynchronize();

		delta_y_pre.release();
		delta_y_next.release();

		if (newton_iteNum < newton_maxIterations) { newton_iteNum++; }
		printf("newton ite num: %d \n jacobi ave_ite num: %lf \n", newton_iteNum, double(jacobi_total_iteNum) / double(newton_iteNum));
		if (jacobi_convergeFlag) { printf("jacobi converge!"); }
		if (newton_convergeFlag) { printf("newton converge!"); }
	}

#ifdef PRECISION_FLOAT
	template class HyperelasticityModule_NewtonMethod<DataType3f>;
#else
	template class HyperelasticityModule_NewtonMethod<DataType3d>;
#endif
}